#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

template <typename T>
__global__ void heat_diffusion_step(T *T_old, T *T_new, int N, int boundary_row, T alpha1, T alpha2)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i > 0 && i < N - 1 && j > 0 && j < N - 1)
    {
        T alpha = (i < boundary_row) ? alpha1 : alpha2;
        T_new[i * N + j] = (1 - alpha) * T_old[i * N + j] +
                           (alpha / 4.0f) * (T_old[(i + 1) * N + j] + T_old[(i - 1) * N + j] +
                                             T_old[i * N + (j + 1)] + T_old[i * N + (j - 1)]);
    }
}

void initialize_grid(float *T, int N, float T_top, float T_other)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            T[i * N + j] = (i == 0) ? T_top : T_other;
        }
    }
}

void save_grid_to_file(float *T, int N, const char *filename)
{
    FILE *file = fopen(filename, "w");
    if (!file)
    {
        perror("Error opening file for writing");
        return;
    }

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            fprintf(file, "%.2f", T[i * N + j]);
            if (j < N - 1)
                fprintf(file, ",");
        }
        fprintf(file, "\n");
    }
    fclose(file);
}

double get_time_diff(struct timespec start, struct timespec end)
{
    return (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1e9;
}

int main(int argc, char *argv[])
{
    if (argc < 8)
    {
        printf("Usage: %s <grid_size> <boundary_row> <alpha1> <alpha2> <iterations> <T_top> <T_other>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int boundary_row = atoi(argv[2]);
    float alpha1 = atof(argv[3]);
    float alpha2 = atof(argv[4]);
    int iterations = atoi(argv[5]);
    float T_top = atof(argv[6]);
    float T_other = atof(argv[7]);

    float *d_T, *d_T_new, *h_T;
    size_t size = N * N * sizeof(float);
    hipMalloc(&d_T, size);
    hipMalloc(&d_T_new, size);
    hipHostMalloc(&h_T, size, hipHostMallocDefault);

    initialize_grid(h_T, N, T_top, T_other);

    dim3 blockSize(16, 16);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    hipMemcpy(d_T, h_T, size, hipMemcpyHostToDevice);

    for (int iter = 0; iter < iterations; iter++)
    {
        heat_diffusion_step<<<gridSize, blockSize>>>(d_T, d_T_new, N, boundary_row, alpha1, alpha2);
        hipDeviceSynchronize();
        float *temp = d_T;
        d_T = d_T_new;
        d_T_new = temp;
    }

    hipMemcpy(h_T, d_T, size, hipMemcpyDeviceToHost);

    clock_gettime(CLOCK_MONOTONIC, &end);
    double elapsed_time = get_time_diff(start, end);

    save_grid_to_file(h_T, N, "heat_output_cuda.csv");

    hipFree(d_T);
    hipFree(d_T_new);
    hipHostFree(h_T);

    printf("CUDA simulation complete. Results saved to heat_output_cuda.csv\n");
    printf("Calculation loop duration: %.6f seconds\n", elapsed_time);

    return 0;
}